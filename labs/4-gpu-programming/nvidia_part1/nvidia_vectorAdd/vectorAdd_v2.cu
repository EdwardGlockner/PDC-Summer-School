
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define DataType double

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
  //@@ Insert code to implement vector addition here
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < len) {
    out[index] = in1[index] + in2[index];
  }  
}

//@@ Insert code to implement timer
struct timeval t_start, t_end;
void myCPUTimer_start(){
  gettimeofday(&t_start, 0);
}
//@@ Insert code to implement timer
void myCPUTimer_stop(){
  hipDeviceSynchronize();
  gettimeofday(&t_end, 0);
  double time = (1000000.0*(t_end.tv_sec-t_start.tv_sec) + t_end.tv_usec-t_start.tv_usec);
  printf("Elasped %6.1f microseconds \n", time);
}

int main(int argc, char **argv) {
  
  int inputLength;
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;


  //@@ Insert code below to read in inputLength from args
  inputLength = atoi(argv[1]);
  printf("The input length is %d\n", inputLength);
  
  //@@ Insert code below to allocate Host memory for input and output
  hostInput1 = (DataType *)malloc(inputLength * sizeof(DataType));
  hostInput2 = (DataType *)malloc(inputLength * sizeof(DataType));
  hostOutput = (DataType *)malloc(inputLength * sizeof(DataType));
  resultRef  = (DataType *)malloc(inputLength * sizeof(DataType));
  
  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
  for(int i=0; i<inputLength; i++){
    hostInput1[i] = 1.0;
    hostInput2[i] = 2.0;
    resultRef[i]  = hostInput1[i] + hostInput2[i];
  }

  //@@ Insert code below to allocate GPU memory here
  hipMalloc((void **)&deviceInput1, inputLength * sizeof(DataType));
  hipMalloc((void **)&deviceInput2, inputLength * sizeof(DataType));
  hipMalloc((void **)&deviceOutput, inputLength * sizeof(DataType));

  //@@ Insert code to below to Copy memory to the GPU here
  hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(DataType), hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(DataType), hipMemcpyHostToDevice);

  //@@ Initialize the 1D grid and block dimensions here
  dim3 blockDim(32);
  dim3 gridDim(ceil(((float)inputLength) / ((float)blockDim.x)));

  //@@ Launch the GPU Kernel here
  vecAdd<<<gridDim, blockDim>>>(deviceInput1, deviceInput2, deviceOutput,
                                inputLength);


  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(DataType), hipMemcpyDeviceToHost);


  //@@ Insert code below to compare the output with the reference
  bool valid = true;
  for(int i=0; i<inputLength; i++){
    if( hostOutput[i] != resultRef[i] ){
      printf("hostOutput[%d] = %f != %f\n", i, hostOutput[i], resultRef[i]);
      valid = false;
    }
  }
  if(valid) printf("valid\n");

  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  //@@ Free the CPU memory here
  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  free(resultRef);

  return 0;
}
